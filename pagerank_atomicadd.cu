#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void initializePagerankArray(float * pagerank_d, int n_vertices) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < n_vertices) {
        pagerank_d[i] = 1.0/(float)n_vertices;
    }
}

__global__ void setPagerankNextArray(float * pagerank_next_d, int n_vertices) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < n_vertices) {
        pagerank_next_d[i] = 0.0;
    }
}


__global__ void addToNextPagerankArray(float * pagerank_d, float * pagerank_next_d, int * n_successors_d, int * successors_d, int * successor_offset_d, float * dangling_value2, int n_vertices) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j;
    int n_suc;
    if (i < n_vertices) {
        n_suc = n_successors_d[i];
        if(n_suc > 0) {
            for(j = 0; j < n_suc; j++) {
                atomicAdd(&(pagerank_next_d[successors_d[successor_offset_d[i]+j]]), 0.85*(pagerank_d[i])/n_suc);
            }
        } else {
            atomicAdd(dangling_value2, 0.85*pagerank_d[i]);
        }
    }
}       

__global__ void finalPagerankArrayForIteration(float * pagerank_next_d, int n_vertices, float dangling_value2) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(i < n_vertices) {
        pagerank_next_d[i] += (dangling_value2 + (1-0.85))/((float)n_vertices);
    }
}

__global__ void setPagerankArrayFromNext(float * pagerank_d, float * pagerank_next_d, int n_vertices, float *diff) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	float temp;
    if(i < n_vertices) {
		temp=pagerank_d[i];
        pagerank_d[i] = pagerank_next_d[i];
        pagerank_next_d[i] = 0.0;
		atomicAdd(diff,((temp - pagerank_d[i])>=0)?(temp- pagerank_d[i]):(pagerank_d[i]-temp) );
    }
}      

int main(int argc, char ** args) {
    if (argc != 2) {
	fprintf(stderr,"Wrong number of args. Provide input graph file.\n");
        exit(-1);
    } 
    hipFree(0);   
    hipError_t err = hipSuccess;
    hipProfilerStart();
    
    // Start CPU timer
    clock_t cycles_to_build, cycles_to_calc;


    // build up the graph
    int i;
    unsigned int n_vertices = 0;
    unsigned int n_edges = 0;
    unsigned int vertex_from = 0, vertex_to = 0, vertex_prev = 0;
    
    // Vertex
    float * pagerank_h, *pagerank_d;
    float *pagerank_next_d;
    int * n_successors_h, *n_successors_d;
    int * successors_h, *successors_d;             
    int * successor_offset_h;
	int *successor_offset_d;

    FILE * fp;
    if ((fp = fopen(args[1], "r")) == NULL) {
        fprintf(stderr,"ERROR: Could not open input file.\n");
        exit(-1);
    }

    // parse input file to count the number of vertices
    while (fscanf(fp, "%u %u", &vertex_from, &vertex_to) != EOF) {
        if (vertex_from > n_vertices) {
            n_vertices = vertex_from;
	}
        else if (vertex_to > n_vertices) {
            n_vertices = vertex_to;
	}
	n_edges++;
    }
    n_vertices++;
     clock_t start = clock();   
    // Allocate flattened data structure host and device memory
    pagerank_h = (float *) malloc(n_vertices * sizeof(*pagerank_h));
    err = hipMalloc((void **)&pagerank_d, n_vertices*sizeof(float));
    err = hipMalloc((void **)&pagerank_next_d, n_vertices*sizeof(float));
    n_successors_h = (int *) calloc(n_vertices, sizeof(*n_successors_h));
    err = hipMalloc((void **)&n_successors_d, n_vertices*sizeof(int));
    successor_offset_h = (int *) malloc(n_vertices * sizeof(*successor_offset_h));
    err = hipMalloc((void **)&successor_offset_d, n_vertices*sizeof(int));

    // Allocate memory for contiguous successors_d data
    successors_h = (int *) malloc(n_edges * sizeof(*successors_h));
    err = hipMalloc((void **)&successors_d, n_edges*sizeof(int));

    // allocate memory for successor pointers
    int offset = 0, edges = 0;      

    // parse input file to count the number of successors of each vertex
    fseek(fp, 0L, SEEK_SET);
    i = 0;
 
    while (fscanf(fp, "%u %u", &vertex_from, &vertex_to) != EOF) {
        n_successors_h[vertex_from] += 1;
	
	// Fill successor_offset_h array
        successor_offset_h[i] = offset;
	if(edges != 0 && vertex_prev != vertex_from) {
	    i = vertex_from;
	    offset = edges;
	    successor_offset_h[i] = offset;	   
	    vertex_prev = vertex_from;
	}

	// Fill successor array
	successors_h[edges] = vertex_to;
	
	edges++;
    }
    successor_offset_h[i] = edges - 1;    

    fclose(fp);

    // Get build time and reset start
    cycles_to_build = clock() - start;
    start = clock();

    // Transfer data structure to the GPU
    err = hipMemcpy(n_successors_d, n_successors_h, n_vertices*sizeof(int), hipMemcpyHostToDevice);
    err = hipMemcpy(successors_d, successors_h, n_edges*sizeof(int), hipMemcpyHostToDevice);
    err = hipMemcpy(successor_offset_d, successor_offset_h, n_vertices*sizeof(int), hipMemcpyHostToDevice);

    // Compute the pagerank
    int n_iterations = 30;
    int iteration = 0;
    int numOfBlocks = 1;                 
    int threadsPerBlock = 1000;                 

    if(n_vertices <= 1024) {
        threadsPerBlock = n_vertices;
        numOfBlocks = 1;
    } else {
        threadsPerBlock = 1024;
        numOfBlocks = (n_vertices + 1023)/1024;   // The "+ 1023" ensures we round up
    }

    float dangling_value_h = 0;
    float dangling_value_h2 = 0;
    float *dangling_value2, *reduced_sums_d;
    int n_blocks = (n_vertices + 2048 - 1)/2048;
    if (n_blocks == 0){
        n_blocks = 1;
    }
    float epsilon = 0.000001;
    float * d_diff;
    float h_diff = epsilon + 1;

    err = hipMalloc((void **)&d_diff, sizeof(float));
    err = hipMalloc((void **)&reduced_sums_d, 1024 * sizeof(float));     
    err = hipMalloc((void **)&dangling_value2, sizeof(float));
    err = hipMemcpy(dangling_value2, &dangling_value_h, sizeof(float), hipMemcpyHostToDevice);

    initializePagerankArray<<<numOfBlocks,threadsPerBlock>>>(pagerank_d, n_vertices);
	hipDeviceSynchronize();
    setPagerankNextArray<<<numOfBlocks,threadsPerBlock>>>(pagerank_next_d, n_vertices);
    hipDeviceSynchronize();
	
    while(epsilon < h_diff && iteration < n_iterations) {  //was 23
       // set the dangling value to 0 
        dangling_value_h = 0;
        err = hipMemcpy(dangling_value2, &dangling_value_h, sizeof(float), hipMemcpyHostToDevice);     
        // initial parallel pagerank_next computation
        addToNextPagerankArray<<<numOfBlocks,threadsPerBlock>>>(pagerank_d, pagerank_next_d, n_successors_d, successors_d, successor_offset_d, dangling_value2, n_vertices);
        hipDeviceSynchronize();
        // get the dangling value
        err = hipMemcpy(&dangling_value_h2, dangling_value2, sizeof(float), hipMemcpyDeviceToHost); 
        // final parallel pagerank_next computation
        finalPagerankArrayForIteration<<<numOfBlocks,threadsPerBlock>>>(pagerank_next_d, n_vertices, dangling_value_h2);
        hipDeviceSynchronize();       
        // Get difference to compare to epsilon
		hipMemset(d_diff, 0, sizeof(float) );
        setPagerankArrayFromNext<<<numOfBlocks,threadsPerBlock>>>(pagerank_d, pagerank_next_d, n_vertices, d_diff);
		hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost);      
		printf("probe2:   %f\n", h_diff);
        hipDeviceSynchronize();

        iteration++;
    }

    err = hipMemcpy(pagerank_h, pagerank_d, n_vertices*sizeof(float), hipMemcpyDeviceToHost);

    // CPU time
    cycles_to_calc = clock() - start;
    int build_milli = cycles_to_build * 1000 / CLOCKS_PER_SEC;
    int calc_milli = cycles_to_calc * 1000 / CLOCKS_PER_SEC;
 
	FILE *f_result;
	f_result=fopen("rg","w");
	for (i=0;i<n_vertices;i++) {
        fprintf(f_result,"Vertex %u:\tpagerank = %.18f\n", i, pagerank_h[i]);
	}  

    printf("Time to build: %d seconds, %d milliseconds\n",build_milli/1000, build_milli%1000);
    printf("Time to calc: %d seconds, %d milliseconds\n",calc_milli/1000, calc_milli%1000);
    printf("iter: %d\n", iteration);


    // Free device global memory
    err = hipFree(pagerank_d);
    err = hipFree(pagerank_next_d);

    // Free host memory
    free(pagerank_h);

    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

