#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>


typedef struct vertex vertex;

struct vertex {
    unsigned int* vertex_id;
    float* pagerank;
    float* pagerank_next;
    unsigned int* n_successors;
    unsigned int** successors;
};


__global__ void initializePageranks(vertex vertices, int n_vertices) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x; 

    if (i < n_vertices) {
        vertices.pagerank[i] = 1.0/(float)n_vertices;
        vertices.pagerank_next[i] = 0.0;
    }
}


__global__ void addToNextPagerank(vertex vertices, float * dangling_value, int n_vertices) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x; 
    int j;

    if(i < n_vertices) {
        if(vertices.n_successors[i] > 0) {
            for(j = 0; j < vertices.n_successors[i]; j++) {
                atomicAdd(&(vertices.pagerank_next[*(vertices.successors[i]+j)]),
				0.85*(vertices.pagerank[i])/vertices.n_successors[i]);
            }
        }else {
            atomicAdd(dangling_value, 0.85*vertices.pagerank[i]);
        }
    }
}

__global__ void finalPagerankForIteration(vertex vertices, int n_vertices, float dangling_value){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i < n_vertices) {
        vertices.pagerank_next[i] += (dangling_value + (1-0.85))/((float)n_vertices);
    }
}

__global__ void setPageranksFromNext(vertex vertices, int n_vertices) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i < n_vertices) {
        vertices.pagerank[i] = vertices.pagerank_next[i];
        vertices.pagerank_next[i] = 0.0;
    }
}

__global__ void convergence(vertex vertices, float * reduced_sums_d, int n_vertices) {
// Each thread computes the diff for two vertexes (thus, half # of blocks needed for this function)
// Because of this, we need to handle the case where only one block is needed
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int i_thr = threadIdx.x;

    __shared__ float sums[1024];                       // blockDim.x == 1024
    
    float temp1, temp2;    
    
    if(i < 1024) {
        reduced_sums_d[i] = 0;
    }

    if(i < n_vertices) {
        temp1 = vertices.pagerank_next[i] - vertices.pagerank[i];
        if(i + (1024 * gridDim.x) < n_vertices) {
            temp2 = vertices.pagerank_next[i+ (1024 * gridDim.x)] - vertices.pagerank[i +(1024*gridDim.x)];
        }else{
            temp2 = 0;
        }

        if(temp1 < 0) {
            temp1 = temp1 * (-1);
        }
        if(temp2 < 0) {
            temp2 = temp2 * (-1);
        }

        sums[i_thr] = temp1 + temp2;
    } else {
        sums[i_thr] = 0;
    }
    __syncthreads();

    int j, index, index2;
    index = i_thr;

    for(j = 0; j < 10; j++) {                    // 10 times as 2^10 = 1024 threads
        if((index+1) % (2 * (1 << j)) == 0) {    // Note: 1 << j == 2^j
            index2 = index - (1 << j);
            sums[index] += sums[index2];
        }
        __syncthreads();
    }

    reduced_sums_d[blockIdx.x] = sums[1023];
}

__global__ void getConvergence(float * reduced_sums_d, float * diff) {
    int j, index, index2;
    index = threadIdx.x;
    
    for(j = 0; j < 10; j++) {                    // 10 times as 2^10 = 1024 threads
        if((index+1) % (2 * (1 << j)) == 0) {    // Note: 1 << j == 2^j
            index2 = index - (1 << j);
            reduced_sums_d[index] += reduced_sums_d[index2];
        }
        __syncthreads();
    }    

    *diff = reduced_sums_d[1023]; 
}


int main(int argc, char ** args) {
    if (argc != 2) {
	fprintf(stderr,"Wrong number of args. Provide input graph file.\n");
        exit(-1);
    } 

    size_t mem_total = 0;
    size_t mem_free = 0;

    hipFree(0); // Initialize the cuda context
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    hipMemGetInfo(&mem_free, &mem_total);
    printf("1. mem_total: %zu, mem_free: %zu\n",mem_total, mem_free);

/*************************************************************************/
    // Start CPU timer
    clock_t cycles_to_build, cycles_to_calc;


/*************************************************************************/
    // build up the graph
    int i;
    unsigned int n_vertices = 0;
    unsigned int vertex_from = 0, vertex_to = 0;

    vertex vertices;

    FILE * fp;
    if ((fp = fopen(args[1], "r")) == NULL) {
        fprintf(stderr,"ERROR: Could not open input file.\n");
        exit(-1);
     }

    // parse input file to count the number of vertices
    // expected format: vertex_from vertex_to
    while (fscanf(fp, "%u %u", &vertex_from, &vertex_to) != EOF) {
        if (vertex_from > n_vertices) {
            n_vertices = vertex_from;
	}
        else if (vertex_to > n_vertices) {
            n_vertices = vertex_to;
	}
    }
    n_vertices++;
    clock_t start = clock();
    // allocate memory for vertices
    //err = cudaMallocManaged((void **)&vertices, n_vertices*sizeof(vertex));
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    err = hipMallocManaged((void **)&vertices.vertex_id, n_vertices*sizeof(vertices.vertex_id));
	err = hipMallocManaged((void **)&vertices.pagerank, n_vertices*sizeof(vertices.pagerank));
	err = hipMallocManaged((void **)&vertices.pagerank_next, n_vertices*sizeof(vertices.pagerank_next));
	err = hipMallocManaged((void **)&vertices.n_successors, n_vertices*sizeof(vertices.n_successors));
	err = hipMallocManaged((void **)&vertices.successors, n_vertices*sizeof(vertices.successors));
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //if (!vertices) {
    //    fprintf(stderr,"Malloc failed for vertices.\n");
    //    exit(-1);
    //}
    //memset((void *)vertices, 0, (size_t)(n_vertices*sizeof(vertex)));
	memset((void *)vertices.vertex_id, 0, (size_t)(n_vertices*sizeof(vertices.vertex_id)));
    memset((void *)vertices.pagerank, 0, (size_t)(n_vertices*sizeof(vertices.pagerank)));
	memset((void *)vertices.pagerank_next, 0, (size_t)(n_vertices*sizeof(vertices.pagerank_next)));
	memset((void *)vertices.n_successors, 0, (size_t)(n_vertices*sizeof(vertices.n_successors)));
	memset((void *)vertices.successors, 0, (size_t)(n_vertices*sizeof(vertices.successors)));
	
    // parse input file to count the number of successors of each vertex
    fseek(fp, 0L, SEEK_SET);
    while (fscanf(fp, "%u %u", &vertex_from, &vertex_to) != EOF) {
        vertices.n_successors[vertex_from]++;
    }
    printf("sizeof(vertex*): %d\n", sizeof(vertex*));
    printf("sizeof(vertex): %d\n", sizeof(vertex));

    hipMemGetInfo(&mem_free, &mem_total);
    printf("mem_total: %zu, mem_free: %zu\n",mem_total, mem_free);
    // allocate memory for successor pointers
    for (i=0; i<n_vertices; i++) {
        vertices.vertex_id[i] = i;
        if (vertices.n_successors[i] > 0) {
            err = hipMallocManaged((void***)&vertices.successors[i],vertices.n_successors[i]*sizeof(unsigned int*));
	    hipMemGetInfo(&mem_free, &mem_total);
	    hipDeviceSynchronize();
    	    //printf("i:%d, mem_total: %zu, mem_free: %zu\n",i, mem_total, mem_free);
            if (!vertices.successors[i]) {
                fprintf(stderr,"hipMallocManaged failed for vertex %d successors (error: %s)\n",i,hipGetErrorString(err));
		hipMemGetInfo(&mem_free, &mem_total);
	    	hipDeviceSynchronize();
    	    	printf("i:%d, mem_total: %zu, mem_free: %zu\n",i, mem_total, mem_free);
                exit(-1);
            }
            memset((void *)vertices.successors[i], 0, (size_t)(vertices.n_successors[i]*sizeof(unsigned int *)));
        }
        else
            vertices.successors[i] = NULL;
    }

    // parse input file to set up the successor pointers
    fseek(fp, 0L, SEEK_SET);
    while (fscanf(fp, "%d %d", &vertex_from, &vertex_to) != EOF) {
        for (i=0; i<vertices.n_successors[vertex_from]; i++) {
            if (*(vertices.successors[vertex_from]+i) == NULL) {
                *(vertices.successors[vertex_from]+i) = vertices.vertex_id[vertex_to];
                break;
            }
            else if (i==vertices.n_successors[vertex_from]-1) {
                printf("Setting up the successor pointers of virtex %u failed",vertex_from);
                return -1;
            }
        }
    }

    fclose(fp);

    // Get time for building data structure
    cycles_to_build = clock() - start;
    int build_msec = cycles_to_build * 1000 / CLOCKS_PER_SEC;
    
    //Reset time 
    start = clock();
   

  /*************************************************************************/
    // compute the pagerank on the GPU
    int n_iterations = 30;
    int numOfBlocks = 1;         // default value for 1000 vertex graph
    int threadsPerBlock = 1000;  // default value for 1000 vertex graph
    int converge_blocks = (n_vertices + 2048 - 1)/2048;
    if(converge_blocks == 0) {
        converge_blocks =1;
    }           


    if(n_vertices <= 1024) {
        threadsPerBlock = n_vertices;
        numOfBlocks = 1;
    } else {
        threadsPerBlock = 1024;
        numOfBlocks = (n_vertices + 1023)/1024;   // The "+ 1023" ensures we round up
    }   


    float dangling_value_h = 0;
    float * dangling_value_d;
    float * reduced_sums_d;
    float epsilon = 0.000001;
    float * d_diff;
    float h_diff = epsilon + 1;

    err = hipMalloc((void **)&d_diff, sizeof(float));
    err = hipMalloc((void **)&reduced_sums_d, 1024 * sizeof(float));

    err = hipMalloc((void **)&dangling_value_d, sizeof(float));
    err = hipMemcpy(dangling_value_d, &dangling_value_h, sizeof(float), hipMemcpyHostToDevice);

    // Initialize pagerank and pagerank_next values
    initializePageranks<<<numOfBlocks,threadsPerBlock>>>(vertices, n_vertices);
    hipDeviceSynchronize(); 
 
    int iteration = 0;
    while(epsilon < h_diff  && iteration < n_iterations) {
        // set the dangling value to 0 
        dangling_value_h = 0;
        err = hipMemcpy(dangling_value_d, &dangling_value_h, sizeof(float), hipMemcpyHostToDevice);
        
        // initial parallel pagerank_next computation
        addToNextPagerank<<<numOfBlocks,threadsPerBlock>>>(vertices, dangling_value_d, n_vertices);

        // get the dangling value
        err = hipMemcpy(&dangling_value_h, dangling_value_d, sizeof(float), hipMemcpyDeviceToHost);
 
        // final parallel pagerank_next computation
        finalPagerankForIteration<<<numOfBlocks,threadsPerBlock>>>(vertices, n_vertices, dangling_value_h);

        convergence<<<converge_blocks, 1024>>>(vertices, reduced_sums_d, n_vertices);
        getConvergence<<<1,1024>>>(reduced_sums_d, d_diff);
        
        // Get difference to compare to epsilon
        hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost);
        
        // Set pagerank = pagerank_next; And then pagerank_next = 0;
        setPageranksFromNext<<<numOfBlocks,threadsPerBlock>>>(vertices, n_vertices);
        
        iteration++;
    }
    hipDeviceSynchronize();
    
    // End CPU Timer
    cycles_to_calc = clock() - start;

    // Print CPU time
    int calc_msec = cycles_to_calc * 1000 / CLOCKS_PER_SEC;

    // print the pagerank values computed on the GPU
    for (i=0;i<n_vertices;i++) {
        printf("AFTER GPU | Vertex %u:\tpagerank = %.6f\n", i, vertices.pagerank[i]);
    }
    
	
    printf("Time to build: %d seconds, %d milliseconds\n", build_msec/1000, build_msec%1000);
    printf("Time to calc: %d seconds, %d milliseconds\n", calc_msec/1000, calc_msec%1000);
    printf("Iteration: %d\n", iteration);


    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

